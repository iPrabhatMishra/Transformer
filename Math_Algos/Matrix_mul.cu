#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) 
{
     __shared__ float sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float sum = 0.0f;
    int numTiles = (N + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < numTiles; ++t) {
        int aCol = t * TILE_WIDTH + threadIdx.x;
        int bRow = t * TILE_WIDTH + threadIdx.y;

        sA[threadIdx.y][threadIdx.x] =
            (row < M && aCol < N) ? A[row * N + aCol] : 0.0f;
        sB[threadIdx.y][threadIdx.x] =
            (bRow < N && col < K) ? B[bRow * K + col] : 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k) {
            sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < K) {
        C[row * K + col] = sum;
    }
}
// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void matrix_multiplication(const float* A, const float* B, float* C, int M, int N, int K) 
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}